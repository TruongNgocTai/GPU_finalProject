#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;
    float elapsed;

    GpuTimer()
    {
        elapsed = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
        float timer;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&timer, start, stop);
        elapsed += timer;
    }

    float Elapsed()
    {
        return elapsed;
    }
};

bool checkCorrect(int * out, int * correctOut, int n)
{
    for (int i = 0; i < n; i++)
        if (out[i] != correctOut[i]) {
            printf("%d\n", i);
            printf("%d >< %d\n", out[i], correctOut[i]);
            return false;
        }
    return true;
}

bool checkCorrectInt32(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
        if (out[i] != correctOut[i]) {
            printf("%d\n", i);
            printf("%d >< %d\n", out[i], correctOut[i]);
            return false;
        }
    return true;
}

// Sequential Radix Sort
void sortByHost(const uint32_t * in, uint32_t * out, int n)
{
	GpuTimer timer; 
    timer.Start();

    int nBits = 4; // Assume: nBits in {1, 2, 4, 8, 16, 32}
    int nBins = 1 << nBits; // 2^nBits

    int * hist = (int *)malloc(nBins * sizeof(int));
    int * histScan = (int *)malloc(nBins * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
    	// Compute histogram
    	memset(hist, 0, nBins * sizeof(int));
    	for (int i = 0; i < n; i++)
    	{
    		int bin = (src[i] >> bit) & (nBins - 1);
    		hist[bin]++;
    	}

    	// Scan histogram (exclusively)
    	memset(histScan, 0, nBins * sizeof(int));
    	for (int bin = 1; bin < nBins; bin++)
    		histScan[bin] = histScan[bin - 1] + hist[bin - 1];

    	// Scatter
    	for (int i = 0; i < n; i++)
    	{
    		int bin = (src[i] >> bit) & (nBins - 1);
    		dst[histScan[bin]] = src[i];
    		histScan[bin]++;
    	}

    	// Swap src and dst
    	uint32_t * temp = src;
    	src = dst;
    	dst = temp;
    }

    // Copy result from src to out
    memcpy(out, src, n * sizeof(uint32_t));

	timer.Stop();
    printf("Time of sortByHost: %.3f ms\n\n", timer.Elapsed());
}

__global__ void computeHist(uint32_t * in, int n, int * hist, int nBins, int bit)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        int bin = (in[idx] >> bit) & (nBins - 1);
        int histIdx = blockIdx.x * nBins + bin;
        atomicAdd(&hist[histIdx], 1);
    }
}

// Compute hist by device
void computeHistByDevice(uint32_t * in, int n, int * hist, int nBins, int bit, int blkSize) 
{
    // Allocate device memories
    uint32_t *d_in;
    int *d_hist;
    int numBlks = (n - 1) / blkSize + 1;
    CHECK(hipMalloc(&d_in, sizeof(uint32_t) * n));
    CHECK(hipMalloc(&d_hist, sizeof(int) * nBins * numBlks));
    
    // Copy data to device memories
    CHECK(hipMemcpy(d_in, in, sizeof(uint32_t) * n, hipMemcpyHostToDevice));

    // Call kernel to scan within each block's input data
    computeHist<<<numBlks, blkSize>>>(d_in, n, d_hist, nBins, bit);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    // Copy result from device memories
    CHECK(hipMemcpy(hist, d_hist, sizeof(int) * nBins * numBlks, hipMemcpyDeviceToHost));
        
    // Free device memories
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_hist));
}

__global__ void scatter(uint32_t * in, uint32_t * out, int n, int * histScan, int nBins, int bit)
{
    int idx = blockIdx.x * blockDim.x;
    if (threadIdx.x == 0) {
        for (int i = 0; i < blockDim.x; i++){
            if (idx + i < n) {
                int bin = (in[idx + i] >> bit) & (nBins - 1);
                int histIdx = blockIdx.x * nBins + bin;
                out[histScan[histIdx]] = in[idx + i];
                histScan[histIdx]++;
            }
        }
    }
}

void scatterByDevice(uint32_t * in, uint32_t * out, int n, int * histScan, int nBins, int bit, int blkSize)
{
    // Allocate device memories
    uint32_t *d_in, *d_out;
    int *d_histScan;
    int numBlks = (n - 1) / blkSize + 1;
    CHECK(hipMalloc(&d_in, sizeof(uint32_t) * n));
    CHECK(hipMalloc(&d_histScan, sizeof(int) * nBins * numBlks));
    CHECK(hipMalloc(&d_out, sizeof(uint32_t) * n));

    // Copy data to device memories
    CHECK(hipMemcpy(d_in, in, sizeof(uint32_t) * n, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_histScan, histScan, sizeof(int) * nBins * numBlks, hipMemcpyHostToDevice));

    // Call kernel to scan within each block's input data
    scatter<<<numBlks, blkSize>>>(d_in, d_out, n, d_histScan, nBins, bit);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    // Copy result from device memories
    CHECK(hipMemcpy(out, d_out, sizeof(uint32_t) * n, hipMemcpyDeviceToHost));
        
    // Free device memories
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_histScan));
    CHECK(hipFree(d_out));
}

// Parallel Radix Sort with k bit
void sortByDeviceLv1(const uint32_t * in, uint32_t * out, int n, int blkSize, int nBits)
{
    GpuTimer timer, computeHistTimer, scanTimer, scatterTimer; 
    timer.Start();

    //int nBits = 4; // Assume: nBits in {1, 2, 4, 8, 16, 32}
    int nBins = 1 << nBits; // 2^nBits

    int numBlks = (n - 1) / blkSize + 1;

    int * hist = (int *)malloc(nBins * numBlks * sizeof(int));
    int * histScan = (int *)malloc(nBins * numBlks * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
    	// Compute histogram
        computeHistTimer.Start();
    	memset(hist, 0, nBins * numBlks * sizeof(int));
        computeHistByDevice(src, n, hist, nBins, bit, blkSize);
        computeHistTimer.Stop();

        scanTimer.Start();
        int curScan = 0;
        for (int binIdx = 0; binIdx < nBins; binIdx++) {
            for (int blkIdx = 0; blkIdx < numBlks; blkIdx++) {
                int histIdx = blkIdx * nBins + binIdx;
                histScan[histIdx] = curScan;
                curScan += hist[histIdx];
            }
        };
        scanTimer.Stop();

    	// Scatter
        scatterTimer.Start();
    	scatterByDevice(src, dst, n, histScan, nBins, bit, blkSize);
        scatterTimer.Stop();

    	// Swap src and dst
    	uint32_t * temp = src;
    	src = dst;
    	dst = temp;
    }

    // Copy result from src to out
    memcpy(out, src, n * sizeof(uint32_t));
    
    timer.Stop();
    printf("Time of sortByDevice Level 1: %.3f ms\n", timer.Elapsed());
    printf("Time of compute hist: %.3f ms\n", computeHistTimer.Elapsed());
    printf("Time of scan hist: %.3f ms\n", scanTimer.Elapsed());
    printf("Time of scatter hist: %.3f ms\n", scatterTimer.Elapsed());
    printf("\n");
}

__global__ void computeHistLocalSort(uint32_t * in, int n, int * hist, int nBits, int bit) 
{
    int nBins = 1 << nBits; // 2^nBits
    int baseIdx = blockDim.x * blockIdx.x;
    int idx = baseIdx + threadIdx.x;

    extern __shared__ uint32_t s_data[];
    int baseCopyIdx = blockDim.x;
    int baseHistIdx = blockDim.x * 2;

    int blockSize;
    if (blockIdx.x == gridDim.x - 1)
        blockSize = n - baseIdx;
    else 
        blockSize = blockDim.x;

    
    if (idx < n) {
        // 1. Input current data to block mem
        s_data[threadIdx.x] = in[idx];
        __syncthreads();
    
        // 2. Local sort
        for (int b = 0; b < nBits; b++) {
            // 2.1. Scan numZerosBefore
            s_data[baseCopyIdx + threadIdx.x] = (s_data[threadIdx.x] >> (bit+b)) & 1;
            __syncthreads();
            
            s_data[baseHistIdx + threadIdx.x] = threadIdx.x == 0 ? 0 : s_data[baseCopyIdx + threadIdx.x - 1];
            __syncthreads();
            for (int stride = 1; stride < blockDim.x; stride *= 2){
                int temp = s_data[baseHistIdx + threadIdx.x];
                if (threadIdx.x >= stride) {
                    temp = s_data[baseHistIdx + threadIdx.x] + s_data[baseHistIdx + threadIdx.x - stride];
                }
                __syncthreads();
                s_data[baseHistIdx + threadIdx.x] = temp ;
                __syncthreads();
            }

            // 2.2. Get numZeros of current block
            int numZeros = blockSize - s_data[baseHistIdx + blockSize-1] - s_data[baseCopyIdx + blockSize-1];

            // 2.3. Calculate rank
            int rank;
            if (s_data[baseCopyIdx + threadIdx.x] == 0)
                rank = threadIdx.x - s_data[baseHistIdx + threadIdx.x];
            if (s_data[baseCopyIdx + threadIdx.x] == 1)
                rank = numZeros + s_data[baseHistIdx + threadIdx.x];
            __syncthreads();

            s_data[baseHistIdx + threadIdx.x] = rank;
            
            // 2.4. Local scatter
            s_data[baseCopyIdx + s_data[baseHistIdx + threadIdx.x]] = s_data[threadIdx.x];
            __syncthreads();
            s_data[threadIdx.x] = s_data[baseCopyIdx + threadIdx.x];
            __syncthreads();
        }

        // 3. Copy value to host
        in[idx] = s_data[threadIdx.x];
        
        // 4. Compute hist
        int bin = (s_data[threadIdx.x] >> bit) & (nBins - 1);
        int histIdx = blockIdx.x * nBins + bin;
        atomicAdd(&hist[histIdx], 1);
    }
}

// Compute hist by device
void computeHistLocalSortByDevice(uint32_t * in, int n, int * hist, int nBits, int bit, int blkSize) 
{
    int nBins = 1 << nBits; // 2^nBits
    // Allocate device memories
    uint32_t *d_in;
    int *d_hist;
    int numBlks = (n - 1) / blkSize + 1;
    int sharedMem = blkSize * (sizeof(uint32_t)*3);
    CHECK(hipMalloc(&d_in, sizeof(uint32_t) * n));
    CHECK(hipMalloc(&d_hist, sizeof(int) * nBins * numBlks));
    
    // Copy data to device memories
    CHECK(hipMemcpy(d_in, in, sizeof(uint32_t) * n, hipMemcpyHostToDevice));

    // Call kernel to scan within each block's input data
    computeHistLocalSort<<<numBlks, blkSize, sharedMem>>>(d_in, n, d_hist, nBits, bit);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    // Copy result from device memories
    CHECK(hipMemcpy(hist, d_hist, sizeof(int) * nBins * numBlks, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(in, d_in, sizeof(uint32_t) * n, hipMemcpyDeviceToHost));
        
    // Free device memories
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_hist));
}

__global__ void scatterLocalSort(uint32_t * in, uint32_t * out, int n, int * hist, int * histScan, int nBins, int bit)
{
    int baseIdx = blockIdx.x * blockDim.x;
    int idx = baseIdx + threadIdx.x;

    extern __shared__ int localHistScan[];
    if (threadIdx.x == 0) {
        localHistScan[0] = 0;
        for (int binIdx = 1; binIdx < nBins; binIdx++) {
            int histIdx = blockIdx.x * nBins + binIdx - 1;
            localHistScan[binIdx] = localHistScan[binIdx - 1] + hist[histIdx]; 
        }
    }
    __syncthreads();

    if (idx < n) {
        int bin = (in[idx] >> bit) & (nBins - 1);
        int histIdx = blockIdx.x * nBins + bin;
        int rank = histScan[histIdx] + threadIdx.x - localHistScan[bin];
        out[rank] = in[idx];
    }
}

void scatterLocalSortByDevice(uint32_t * in, uint32_t * out, int n, int * hist, int * histScan, int nBins, int bit, int blkSize)
{
    // Allocate device memories
    uint32_t *d_in, *d_out;
    int *d_histScan, *d_hist;
    int numBlks = (n - 1) / blkSize + 1;
    CHECK(hipMalloc(&d_in, sizeof(uint32_t) * n));
    CHECK(hipMalloc(&d_out, sizeof(uint32_t) * n));
    CHECK(hipMalloc(&d_histScan, sizeof(int) * nBins * numBlks));
    CHECK(hipMalloc(&d_hist, sizeof(int) * nBins * numBlks));

    // Copy data to device memories
    CHECK(hipMemcpy(d_in, in, sizeof(uint32_t) * n, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_histScan, histScan, sizeof(int) * nBins * numBlks, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_hist, hist, sizeof(int) * nBins * numBlks, hipMemcpyHostToDevice));

    // Call kernel to scan within each block's input data
    scatterLocalSort<<<numBlks, blkSize, nBins * sizeof(int)>>>(d_in, d_out, n, d_hist, d_histScan, nBins, bit);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    // Copy result from device memories
    CHECK(hipMemcpy(out, d_out, sizeof(uint32_t) * n, hipMemcpyDeviceToHost));
        
    // Free device memories
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    CHECK(hipFree(d_hist));
    CHECK(hipFree(d_histScan));
}

// Parallel Radix Sort with k bit
void sortByDeviceLv2(const uint32_t * in, uint32_t * out, int n, int blkSize, int nBits)
{
	GpuTimer timer, computeHistTimer, scanTimer, scatterTimer; 
    timer.Start();

    //int nBits = 4; // Assume: nBits in {1, 2, 4, 8, 16, 32}
    int nBins = 1 << nBits; // 2^nBits

    int numBlks = (n - 1) / blkSize + 1;

    int * hist = (int *)malloc(nBins * numBlks * sizeof(int));
    int * histScan = (int *)malloc(nBins * numBlks * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
    	// Compute histogram
        computeHistTimer.Start();
    	memset(hist, 0, nBins * numBlks * sizeof(int));
        computeHistLocalSortByDevice(src, n, hist, nBits, bit, blkSize);
        computeHistTimer.Stop();

        scanTimer.Start();
        int curScan = 0;
        for (int binIdx = 0; binIdx < nBins; binIdx++) {
            for (int blkIdx = 0; blkIdx < numBlks; blkIdx++) {
                int histIdx = blkIdx * nBins + binIdx;
                histScan[histIdx] = curScan;
                curScan += hist[histIdx];
            }
        }
        scanTimer.Stop();

        // Scatter
        scatterTimer.Start();
    	scatterLocalSortByDevice(src, dst, n, hist, histScan, nBins, bit, blkSize);
        scatterTimer.Stop();

    	// Swap src and dst
    	uint32_t * temp = src;
    	src = dst;
    	dst = temp;
    }

    // Copy result from src to out
    memcpy(out, src, n * sizeof(uint32_t));
    
    timer.Stop();
    printf("Time of sortByDevice Level 2: %.3f ms\n", timer.Elapsed());
    printf("Time of compute hist: %.3f ms\n", computeHistTimer.Elapsed());
    printf("Time of scan hist: %.3f ms\n", scanTimer.Elapsed());
    printf("Time of scatter hist: %.3f ms\n", scatterTimer.Elapsed());
    printf("\n");
}

// Parallel Radix Sort with k = 1 bit
void sortByDevice(const uint32_t * in, uint32_t * out, int n, int blockSize)
{
	GpuTimer timer; 
    timer.Start();
	
	// TODO
    thrust::device_vector<uint32_t> dv_out(in, in + n);
    thrust::sort(dv_out.begin(), dv_out.end());
    thrust::copy(dv_out.begin(), dv_out.end(), out);
    
	timer.Stop();
    printf("Time of sortByDevice by thrust: %.3f ms\n\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = (1 << 24) + 1;
    printf("Input size: %d\n\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
        in[i] = rand();

    // DETERMINE BLOCK SIZE
    int blockSize = 512; // Default 
    int nBits = 4;
    if (argc >= 2)
        blockSize = atoi(argv[1]);

    if (argc == 3)
        nBits = atoi(argv[2]);

    // SORT BY HOST
    sortByHost(in, correctOut, n);
    
    // SORT BY DEVICE LEVEL 1
    sortByDeviceLv1(in, out, n, blockSize, nBits);
    if (checkCorrectInt32(out, correctOut, n) == false)
        printf("sortByDevice is INCORRECT!\n\n");

    // SORT BY DEVICE LEVEL 2
    sortByDeviceLv2(in, out, n, blockSize, nBits);
    if (checkCorrectInt32(out, correctOut, n) == false)
        printf("sortByDevice is INCORRECT!\n\n");
        
    // SORT BY DEVICE by THRUST
    sortByDevice(in, out, n, blockSize);
    if (checkCorrectInt32(out, correctOut, n) == false)
    	printf("sortByDevice is INCORRECT!\n\n");

    // FREE MEMORIES
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}